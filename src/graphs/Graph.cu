#include "hip/hip_runtime.h"
#include "graphs/Graph.cuh"
#include "config/config.h"
#include <filesystem>

Graph::Graph(const float ws)
{
    if(VERBOSE)
        {
            printf("/***************************/\n");
            printf("/* Grid Dimension: %d */\n", W_DIM + C_DIM + V_DIM);
            printf("/***************************/\n");
        }

    h_numPartialSums_ = iDivUp(NUM_R1_REGIONS, h_blockSize_);

    d_validCounterArray_     = thrust::device_vector<int>(NUM_R1_REGIONS);
    d_counterArray_          = thrust::device_vector<int>(NUM_R1_REGIONS);
    d_vertexScoreArray_      = thrust::device_vector<float>(NUM_R1_REGIONS);
    d_activeVerticesScanIdx_ = thrust::device_vector<int>(NUM_R1_REGIONS);
    d_activeSubVertices_     = thrust::device_vector<int>(NUM_R2_REGIONS);
    d_minValueInRegion_      = thrust::device_vector<float>(NUM_R1_REGIONS * STATE_DIM);
    d_partialSums_           = thrust::device_vector<float>(h_numPartialSums_);
    d_totalScore_            = thrust::device_vector<float>(1, 0.0);

    d_validCounterArray_ptr_ = thrust::raw_pointer_cast(d_validCounterArray_.data());
    d_counterArray_ptr_      = thrust::raw_pointer_cast(d_counterArray_.data());
    d_vertexScoreArray_ptr_  = thrust::raw_pointer_cast(d_vertexScoreArray_.data());
    d_activeSubVertices_ptr_ = thrust::raw_pointer_cast(d_activeSubVertices_.data());
    d_minValueInRegion_ptr_  = thrust::raw_pointer_cast(d_minValueInRegion_.data());
    d_partialSums_ptr_       = thrust::raw_pointer_cast(d_partialSums_.data());
    d_totalScore_ptr_        = thrust::raw_pointer_cast(d_totalScore_.data());

    initializeRegions();

    std::ostringstream filename;
    std::filesystem::create_directories("Data");
    std::filesystem::create_directories("Data/RegionMins");

    filename.str("");
    filename << "Data/RegionMins/RegionMins_" << ws << ".csv";
    copyAndWriteVectorToCSV(d_minValueInRegion_, filename.str(), NUM_R1_REGIONS, 1, false);
}

void Graph::initializeRegions()
{
    initializeRegions_kernel<<<iDivUp(NUM_R1_REGIONS, h_blockSize_), h_blockSize_>>>(d_minValueInRegion_ptr_);
}

/***************************/
/* INITIALIZE REGIONS KERNEL */
/***************************/
// --- one thread per R1 region ---
__global__ void initializeRegions_kernel(float* minValueInRegion)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= NUM_R1_REGIONS) return;

    int wRegion = tid % (W_R1_LENGTH * W_R1_LENGTH * W_R1_LENGTH);
    int wIndex[W_DIM];
    int temp = wRegion;
    for(int i = W_DIM - 1; i >= 0; --i)
        {
            wIndex[i] = temp % W_R1_LENGTH;
            temp /= W_R1_LENGTH;
        }

    for(int i = 0; i < W_DIM; ++i)
        {
            minValueInRegion[tid * STATE_DIM + i] = W_MIN + wIndex[i] * W_R1_SIZE;
        }

    int aRegion = (tid / (W_R1_LENGTH * W_R1_LENGTH * W_R1_LENGTH)) % (C_R1_LENGTH * C_R1_LENGTH);
    int aIndex[C_DIM];
    temp = aRegion;
    for(int i = C_DIM - 1; i >= 0; --i)
        {
            aIndex[i] = temp % C_R1_LENGTH;
            temp /= C_R1_LENGTH;
        }
    for(int i = 0; i < C_DIM; ++i)
        {
            minValueInRegion[tid * STATE_DIM + W_DIM + i] = C_MIN + aIndex[i] * C_R1_SIZE;
        }

    int vRegion = (tid / (W_R1_LENGTH * W_R1_LENGTH * W_R1_LENGTH * C_R1_LENGTH * C_R1_LENGTH)) % V_R1_LENGTH;
    int vIndex[V_DIM];
    temp = vRegion;
    for(int i = V_DIM - 1; i >= 0; --i)
        {
            vIndex[i] = temp % V_R1_LENGTH;
            temp /= V_R1_LENGTH;
        }
    for(int i = 0; i < V_DIM; ++i)
        {
            minValueInRegion[tid * STATE_DIM + W_DIM + C_DIM + i] = V_MIN + vIndex[i] * V_R1_SIZE;
        }
}

__host__ __device__ int getRegion(float* coord)
{
    // --- Workspace ---
    int wRegion = 0;
    int factor  = 1;
    int index;
    for(int i = W_DIM - 1; i >= 0; --i)
        {
            index = (int)(W_R1_LENGTH * (coord[i] - W_MIN) / (W_MAX - W_MIN));
            if(index >= W_R1_LENGTH) index = W_R1_LENGTH - 1;
            if(index < 0) index = 0;

            wRegion += factor * index;
            factor *= W_R1_LENGTH;
        }

    if(V_DIM == 1 && C_DIM == 1)
        {
            return wRegion;
        }

    // --- Attitude ---
    int aRegion = 0;
    if(C_R1_LENGTH > 1)
        {
            factor = 1;
            for(int i = C_DIM - 1; i >= 0; --i)
                {
                    index = (int)(C_R1_LENGTH * (coord[i + W_DIM] - C_MIN) / (C_MAX - C_MIN));
                    if(index >= C_R1_LENGTH) index = C_R1_LENGTH - 1;
                    if(index < 0) index = 0;

                    aRegion += factor * index;
                    factor *= C_R1_LENGTH;
                }
        }

    // --- Velocity ---
    int vRegion = 0;
    if(V_R1_LENGTH > 1)
        {
            factor = 1;
            for(int i = V_DIM - 1; i >= 0; --i)
                {
                    index = (int)(V_R1_LENGTH * (coord[i + W_DIM + C_DIM] - V_MIN) / (V_MAX - V_MIN));
                    if(index >= V_R1_LENGTH) index = V_R1_LENGTH - 1;
                    if(index < 0) index = 0;

                    vRegion += factor * index;
                    factor *= V_R1_LENGTH;
                }
        }

    return wRegion * pow(C_R1_LENGTH, C_DIM) * pow(V_R1_LENGTH, V_DIM) + aRegion * pow(V_R1_LENGTH, V_DIM) + vRegion;
}

__device__ int getSubRegion(float* coord, int r1, float* minRegion)
{
    // --- Workspace ---
    int wRegion = 0;
    int factor  = 1;
    int index;

    for(int i = W_DIM - 1; i >= 0; --i)
        {
            index = (int)(W_R2_LENGTH * (coord[i] - minRegion[r1 * STATE_DIM + i]) / (W_R1_SIZE));
            if(index >= W_R2_LENGTH) index = W_R2_LENGTH - 1;
            if(index < 0) index = 0;

            wRegion += factor * index;
            factor *= W_R2_LENGTH;
        }

    // --- Attitude ---
    int aRegion = 0;
    if(C_R2_LENGTH > 1)
        {
            factor = 1;
            for(int i = C_DIM - 1; i >= 0; --i)
                {
                    index = (int)(C_R2_LENGTH * (coord[i + W_DIM] - minRegion[r1 * STATE_DIM + i + W_DIM]) / (C_R1_SIZE));
                    if(index >= C_R2_LENGTH) index = C_R2_LENGTH - 1;
                    if(index < 0) index = 0;

                    aRegion += factor * index;
                    factor *= C_R2_LENGTH;
                }
        }

    // --- Velocity ---
    int vRegion = 0;
    if(V_R2_LENGTH > 1)
        {
            factor = 1;
            for(int i = V_DIM - 1; i >= 0; --i)
                {
                    index = (int)(V_R2_LENGTH * (coord[i + W_DIM + C_DIM] - minRegion[r1 * STATE_DIM + i + W_DIM + C_DIM]) / (V_R1_SIZE));
                    if(index >= V_R2_LENGTH) index = V_R2_LENGTH - 1;
                    if(index < 0) index = 0;

                    vRegion += factor * index;
                    factor *= V_R2_LENGTH;
                }
        }

    return r1 * NUM_R2_PER_R1 + (wRegion * pow(C_R2_LENGTH, C_DIM) * pow(V_R2_LENGTH, V_DIM) + aRegion * pow(V_R2_LENGTH, V_DIM) + vRegion);
}

void Graph::updateVertices()
{
    if(NUM_R1_REGIONS > 1024)
        {
            // --- Update R1 Scores ---
            partialReduction_kernel<<<h_numPartialSums_, h_blockSize_>>>(d_activeSubVertices_ptr_, d_validCounterArray_ptr_,
                                                                         d_counterArray_ptr_, d_vertexScoreArray_ptr_, d_partialSums_ptr_);
            // --- Sum R1 Scores ---
            globalReduction_kernel<<<1, h_numPartialSums_>>>(d_partialSums_ptr_, d_totalScore_ptr_, h_numPartialSums_);

            // --- Normalize R1 Scores ---
            updateSampleAcceptance_kernel<<<h_numPartialSums_, h_blockSize_>>>(d_validCounterArray_ptr_, d_vertexScoreArray_ptr_,
                                                                               d_totalScore_ptr_);
        }
    else
        {
            // --- Update vertex scores and sampleScoreThreshold ---
            updateVertices_kernel<<<1, NUM_R1_REGIONS>>>(d_activeSubVertices_ptr_, d_validCounterArray_ptr_, d_counterArray_ptr_,
                                                         d_vertexScoreArray_ptr_);
        }
}

/***************************/
/* PARTIAL REDUCTION KERNEL */
/***************************/
// --- calculates score for each region and does a partial blockwise sum of scores. ---
__global__ void
partialReduction_kernel(int* activeSubVertices, int* validCounterArray, int* counterArray, float* vertexScores, float* partialSums)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= NUM_R1_REGIONS) return;

    float score = 0.0;

    if(validCounterArray[tid] > 0)
        {
            int numValidSamples = validCounterArray[tid];
            float coverage      = 0;

            // --- Thread loops through all sub vertices to determine vertex coverage. ---
            for(int i = tid * NUM_R2_PER_R1; i < (tid + 1) * NUM_R2_PER_R1; ++i)
                {
                    coverage += activeSubVertices[i];
                }
            coverage /= NUM_R2_PER_R1;

            // --- From OMPL Syclop ref: https://ompl.kavrakilab.org/classompl_1_1control_1_1Syclop.html---
            float freeVol = (EPSILON + numValidSamples) / (EPSILON + numValidSamples + (counterArray[tid] - numValidSamples)) * W_R1_VOL;
            score         = pow(freeVol, 4) / ((1 + coverage) * (1 + pow(counterArray[tid], 2)));
            vertexScores[tid] = score;
        }

    // --- Sum scores from each thread to determine score threshold ---
    typedef hipcub::BlockReduce<float, NUM_PARTIAL_SUMS> BlockReduceFloatT;
    __shared__ typename BlockReduceFloatT::TempStorage tempStorageFloat;
    float blockSum = BlockReduceFloatT(tempStorageFloat).Sum(score);

    if(threadIdx.x == 0)
        {
            partialSums[threadIdx.x] = blockSum;
        }
}

/***************************/
/* GLOBAL REDUCTION KERNEL */
/***************************/
// --- Sums all partial sums into totalScore ---
__global__ void globalReduction_kernel(float* partialSums, float* totalScore, int numPartialSums)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= numPartialSums) return;

    typedef hipcub::BlockReduce<float, NUM_PARTIAL_SUMS> BlockReduceFloatT;
    __shared__ typename BlockReduceFloatT::TempStorage tempStorageFloat;
    float blockSum = BlockReduceFloatT(tempStorageFloat).Sum(partialSums[tid]);

    if(threadIdx.x == 0)
        {
            atomicAdd(totalScore, blockSum);
        }
}

/***************************/
/* UPDATE SAMPLE ACCEPTANCE KERNEL */
/***************************/
// --- normalizes score for each active region ---
__global__ void updateSampleAcceptance_kernel(int* validCounterArray, float* vertexScores, float* totalScore)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= NUM_R1_REGIONS) return;
    if(validCounterArray[tid] == 0)
        {
            vertexScores[tid] = 1.0f;
        }
    else
        {
            vertexScores[tid] = EPSILON + (vertexScores[tid] / *totalScore);
        }
}

/***************************/
/* VERTICES UPDATE KERNEL  */
/***************************/
// --- Updates Vertex Scores for device graph vectors. Determines new threshold score for future samples in expansion set. ---
__global__ void updateVertices_kernel(int* activeSubVertices, int* validCounterArray, int* counterArray, float* vertexScores)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= NUM_R1_REGIONS - 1) return;

    __shared__ float s_totalScore;
    float score = 0.0;

    if(validCounterArray[tid] > 0)
        {
            int numValidSamples = validCounterArray[tid];
            float coverage      = 0;

            // --- Thread loops through all sub vertices to determine vertex coverage. ---
            for(int i = tid * NUM_R2_PER_R1; i < (tid + 1) * NUM_R2_PER_R1; ++i)
                {
                    coverage += activeSubVertices[i];
                }

            coverage /= NUM_R2_PER_R1;

            // --- From OMPL Syclop ref: https://ompl.kavrakilab.org/classompl_1_1control_1_1Syclop.html---
            float freeVol = (EPSILON + numValidSamples) / (EPSILON + numValidSamples + (counterArray[tid] - numValidSamples)) * W_R1_VOL;
            score         = pow(freeVol, 4) / ((1 + coverage) * (1 + pow(counterArray[tid], 2)));
        }

    // --- Sum scores from each thread to determine score threshold ---
    typedef hipcub::BlockReduce<float, NUM_R1_REGIONS_KERNEL1> BlockReduceFloatT;
    __shared__ typename BlockReduceFloatT::TempStorage tempStorageFloat;
    float blockSum = BlockReduceFloatT(tempStorageFloat).Sum(score);

    if(threadIdx.x == 0)
        {
            s_totalScore = blockSum;
        }
    __syncthreads();

    // --- Update vertex scores ---
    if(validCounterArray[tid] == 0)
        {
            vertexScores[tid] = 1.0f;
        }
    else
        {
            // TODO: check if adding epsilon is ok.
            vertexScores[tid] = EPSILON + (score / s_totalScore);
        }
}